/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	int numThreads = blockDim.x * gridDim.x;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	// If 19 nodes & 9 threads, each thread gets 3
	int nodesPerThread = (int)((double)(numNodes / numThreads) + 1);

	// If more threads than nodes, make some idled threads
	if (nodesPerThread == 0) {
		nodesPerThread = 1;
	}

	// Calculate start and end nodes for this thread
	int i = tid * nodesPerThread;
	int max = i + nodesPerThread;
	if (max > numNodes) {
		max = numNodes;
	}

	// Iterate through select nodes
	while (i < max) {
		// If already matched
		if (matches[i] > -1) {
			continue;
		}

		// Get i's strongNeighbor, j
		int j = strongNeighbor[i];
		// If j's strongNeighbor is i
		if (strongNeighbor[j] == i) {
			matches[i] = j;
			// we have a synchronization problem here
			// another thread may do this at same time
			// wastes GPU, no memory issues
			matches[j] = i;
		}
	}

	return;
}
