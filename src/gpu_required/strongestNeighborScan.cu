/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// http://users.wfu.edu/choss/CUDA/docs/Lecture%205.pdf

__global__ void strongestNeighborScan_gpu
(
	int * src, 
	int * oldDst, int * newDst, 
	int * oldWeight, int * newWeight, 
	int * madeChanges, int distance, int numEdges
) {
	int numThreads = blockDim.x * gridDim.x;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	// If 19 edges & 9 threads, each thread gets 3
	int edgesPerThread = (int)((double)(numEdges / numThreads) + 1);

	// If more threads than edges, make some idled threads
	if (edgesPerThread == 0) {
		edgesPerThread = 1;
	}

	// Calculate start and end edge for this thread
	int i = tid * edgesPerThread;
	int max = i + edgesPerThread;
	if (max > numEdges) {
		max = numEdges;
	}

	// Iterate through each edge
	while (i < max) {
		// if L is unavailable
		if (i - distance < 0 || src[i] != src[i-distance]) {
			goto rBigger;
		}

		// if R is bigger
		if (oldWeight[i] > oldWeight[i-distance]) {
			goto rBigger;
		}

		/* goto locations */
		/*lBigger:*/ // if oldDst[i-distance] is larger
			newDst[i] = oldDst[i-distance];
			newWeight[i] = oldWeight[i-distance];
			*madeChanges = 1;
			++i;
			continue;

		rBigger: // if oldDst[i] is larger
			newDst[i] = oldDst[i];
			newWeight[i] = oldWeight[i];
			++i;
			continue;
	}

	return;
}
