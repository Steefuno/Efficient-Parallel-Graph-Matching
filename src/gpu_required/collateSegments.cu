/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu
(
	int * src, int * scanResult, int * output, int numEdges
) {
	int src_i = 0;
	int output_i = 0;

	// check each in src with next
	while (src_i < numEdges-1) {
		if (src[src_i] != src[src_i+1]) {
			output[output_i++] = scanResult[src_i];
		}
		src_i++;
	}
	// insert last in src
	output[output_i] = scanResult[src_i];

	return;
}
